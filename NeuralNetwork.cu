#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include "NeuralNetwork.cudah"
#include "Random.cu"
#include <chrono>

__global__ void feedForwardLayer(float * values, float * biases, float * weights, int * dims, int * sumDims, int * sumWeights, int layer) {
    /* This gets the 'index' of the thread. The thread id is the id of the thread in the thread block
       The blockDim is the amount of threads per block and the block id is the id of the block this thread is in
       This basically gives the index of the thread so you can use it to access memory.
    */
    const int neuron = blockDim.x * blockIdx.x + threadIdx.x;

    // If the thread index is less than the size of the array.
    if(neuron < dims[layer]) {
        for(int subNeuron = sumDims[layer-1]; subNeuron < sumDims[layer]; subNeuron++) {
            values[sumDims[layer] + neuron] += values[subNeuron] * weights[sumWeights[layer-1] + dims[layer]*(subNeuron-sumDims[layer-1]) + neuron];
        }
        values[sumDims[layer] + neuron] += biases[sumDims[layer] + neuron];
    }
};

__global__ void getBackPropDeltas(float * deltaBiases, float * deltaWeights, float * values, float * weights, int * dims, int * sumDims, int * sumWeights, int layer) {
    /* This gets the 'index' of the thread. The thread id is the id of the thread in the thread block
       The blockDim is the amount of threads per block and the block id is the id of the block this thread is in
       This basically gives the index of the thread so you can use it to access memory.
    */
    // In this case, the neuron is the same value as the neuron in the iterator on the CPU
    int neuron = blockDim.x * blockIdx.x + threadIdx.x;

    // If the thread index is less than the size of the array.
    if(neuron < dims[layer]) {
        deltaBiases[neuron] = 0;
        for(int subNeuron = 0; subNeuron < dims[layer-1]; subNeuron++) {
            deltaWeights[sumWeights[layer-1] + dims[layer] * (subNeuron) + neuron] = values[sumDims[layer-1] + subNeuron] * deltaBiases[sumDims[layer] + neuron];
        }
    }
    // The reason for the separation is because you don't want multiple threads to change the same value at the same time.
    int subNeuron = neuron;
    if(subNeuron < dims[layer-1]) {
        for(int neuron = 0; neuron < dims[layer]; neuron++) {
            deltaBiases[sumDims[layer-1] + subNeuron] += weights[sumWeights[layer-1] + dims[layer] * (subNeuron) + neuron] * deltaBiases[sumDims[layer] + neuron];
        }
    }
};

__global__ void applyDeltas(float * deltaBiases, float * biases, float * deltaWeights, float * weights, int totalNNSize, int totalWeights, float learningRate) {
    /* This gets the 'index' of the thread. The thread id is the id of the thread in the thread block
       The blockDim is the amount of threads per block and the block id is the id of the block this thread is in
       This basically gives the index of the thread so you can use it to access memory.
    */
    int index = blockDim.x * blockIdx.x + threadIdx.x;


    if(index < totalWeights) {
        weights[index] += deltaWeights[index] * learningRate;
    }
    // If the thread index is less than the size of the array.
    if(index < totalNNSize) {
        biases[index] += deltaBiases[index] * learningRate;
    }
};

Network::Network() {
    read();
};

Network::Network(int dimsLen, int * neuronCountList, float learningRateInput) {
    // Set the learning rate
    learningRate = learningRateInput;
    // Sets dims and dimsLength
    dimsLength = dimsLen;
    dims = (int *) malloc(dimsLength * sizeof(int));
    for(int layer = 0; layer < dimsLength; layer++) {
        dims[layer] = neuronCountList[layer];
        std::<< "Dims[" << layer << "]: " << dims[layer] << std::endl;
    }
    printf("Dimsptr: %p\n", dims);
    initAll();
};

void Network::initAll() {
    // Gets the total size of the network.
    totalNNSize = 0;
    for(int layer = 0; layer < dimsLength; layer++) {
        totalNNSize += dims[layer];
    }
    // Gets the total amount of weights, and the amount of weights per layer
    weightDims = (int *) calloc(dimsLength, sizeof(int));
    weightDims[dimsLength-1] = 0;
    totalWeights = 0;
    for(int layer = 0; layer < dimsLength-1; layer++) {
        weightDims[layer] = dims[layer] * dims[layer+1];
        totalWeights += weightDims[layer];
    }

    // Gets the amount of weights from the 0th weight for each layer
    sumWeights = (int *) malloc((dimsLength) * sizeof(int));
    sumWeights[0] = 0;
    sumWeights[1] = weightDims[0];
    for(int layer = 2; layer < dimsLength; layer++) {
        sumWeights[layer] = sumWeights[layer-1] + weightDims[layer-1];
    }

    // Gets the amount of neuron from the 0th neuron for each layer
    sumDims = (int *) malloc((dimsLength+1) * sizeof(int));
    sumDims[0] = 0;
    sumDims[1] = dims[0];
    for(int layer = 2; layer < dimsLength+1; layer++) {
        sumDims[layer] = sumDims[layer-1] + dims[layer-1];
    }

    // Creates a new, empty value array.
    values = (float *) calloc(totalNNSize, sizeof(float));
    // Makes the delta variables for backpropagation
    deltaBiases = (float *) calloc(totalNNSize, sizeof(float));
    deltaWeights = (float *) calloc(totalWeights, sizeof(float));
    // Initializes the weights and biases with a value of 0.
    biases = (float *) calloc(totalNNSize, sizeof(float));
    weights = (float *) calloc(totalWeights, sizeof(float));
};

void Network::randomize() {
    srand(totalWeights);
    std::cout << "Randomizing biases... 0/" << totalNNSize <<std::endl;
    for(int neuron = 0; neuron < totalNNSize; neuron++) {
        biases[neuron] = (2 * (float(rand()) / (float) RAND_MAX) - 1);
    }

    std::cout << "Randomizing weights... 0/" << totalWeights <<std::endl;
    for(int weight = 0; weight < totalWeights; weight++) {
        weights[weight] = (2*rand()/RAND_MAX)-1;
    }
};

void Network::loadNetworkToGPU() {
    // These two do not need memcpy because they are initialized on the GPU.
    hipMalloc(&deviceDeltaBiases, totalNNSize * sizeof(float));
    hipMalloc(&deviceDeltaWeights, totalWeights * sizeof(float));

    hipMalloc(&deviceBiases, totalNNSize * sizeof(float));
    hipMalloc(&deviceWeights, totalWeights * sizeof(float));
    hipMalloc(&deviceDims, dimsLength * sizeof(int));
    hipMalloc(&deviceSumDims, (dimsLength+1) * sizeof(int));
    hipMalloc(&deviceSumWeights, (dimsLength) * sizeof(int));
    hipMalloc(&deviceValues, totalNNSize * sizeof(float));

    hipMemcpy(deviceBiases, biases, totalNNSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deviceWeights, weights, totalWeights * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deviceDims, dims, dimsLength * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceSumDims, sumDims, (dimsLength+1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceSumWeights, sumWeights, (dimsLength) * sizeof(int), hipMemcpyHostToDevice);
    loaded = true;
};

float * Network::feedForwardGPU(float input[]) {
    // Creates a new, empty value array.
    for(int i = 0; i < totalNNSize; i++) {
        values[i] = 0;
    }

    // Puts the inputs into the values to be fed forward.
    for(int neuron = 0; neuron < dims[0]; neuron++) {
        values[neuron] = input[neuron];
    }

    // Allocates device memory for the values
    hipMemcpy(deviceValues, values, totalNNSize * sizeof(float), hipMemcpyHostToDevice);
    // For each layer, feed forward using a GPU thread instead of a for loop.
    for(int layer = 1; layer < dimsLength; layer++) {
        int dimGrid = (dims[layer] + THREADSPERBLOCK - 1)/THREADSPERBLOCK;
        feedForwardLayer <<< dimGrid, THREADSPERBLOCK >>> (deviceValues, deviceBiases, deviceWeights, deviceDims, deviceSumDims, deviceSumWeights, layer);
    }
    hipMemcpy(values, deviceValues, totalNNSize * sizeof(float), hipMemcpyDeviceToHost);

    float * lastLayer = (float *) malloc(sizeof(float) * dims[dimsLength-1]);
    for(int neuron = sumDims[dimsLength-1]; neuron < totalNNSize; neuron++) {
        lastLayer[neuron - sumDims[dimsLength-1]] = values[neuron];
    }
    return lastLayer;
};

void Network::backPropGPU(float expected[], float * result) {
    float * deltaBiasesTemp = (float *) calloc(totalNNSize, sizeof(float));
    for(int i = 0; i < dims[dimsLength-1]; i++) {;
        deltaBiasesTemp[sumDims[dimsLength-1] + i] = expected[i] - result[i];
    }
    hipMemcpy(deviceDeltaBiases, deltaBiasesTemp, totalNNSize * sizeof(float), hipMemcpyHostToDevice);
    int dimGrid;
    for(int layer = dimsLength-1; layer > 0; layer--) {
        int amountOfNeurons = dims[layer] > dims[layer-1] ? dims[layer] : dims[layer-1];
        dimGrid = (amountOfNeurons + THREADSPERBLOCK - 1)/THREADSPERBLOCK;
        getBackPropDeltas <<< dimGrid, THREADSPERBLOCK >>> (deviceDeltaBiases, deviceDeltaWeights, deviceValues, deviceWeights, deviceDims, deviceSumDims, deviceSumWeights, layer);
    }
    // dimGrid = (totalWeights + THREADSPERBLOCK - 1)/THREADSPERBLOCK;
    // applyDeltas <<< dimGrid, THREADSPERBLOCK >>> (deviceDeltaBiases, deviceBiases, deviceDeltaWeights, deviceWeights, totalNNSize, totalWeights, learningRate);
}

float * Network::feedForwardCPU(float input[]) {
    // Puts the inputs into the values to be fed forward.
    for(int neuron = 0; neuron < dims[0]; neuron++) {
        values[neuron] = input[neuron];
    }
    for(int layer = 1; layer < dimsLength; layer++) {
        // This for loop wouldn't exist if this was for the GPU since each GPU thread gets a neuron.
        for(int neuron = 0; neuron < dims[layer]; neuron++) {
            for(int subNeuron = sumDims[layer-1]; subNeuron < sumDims[layer]; subNeuron++) {
                values[sumDims[layer] + neuron] += values[subNeuron] * weights[sumWeights[layer-1] + dims[layer]*(subNeuron-sumDims[layer-1]) + neuron];
            }
            values[sumDims[layer] + neuron] += biases[sumDims[layer] + neuron];
        }
    }
    float * lastLayer = (float *) malloc(sizeof(float) * dims[dimsLength-1]);
    for(int neuron = sumDims[dimsLength-1]; neuron < totalNNSize; neuron++) {
        lastLayer[neuron - sumDims[dimsLength-1]] = values[neuron];
    }
    return lastLayer;
};

void Network::getBackPropDeltasCPU(float expected[], float * result) {
    for(int i = 0; i < dims[dimsLength-1]; i++) {
        deltaBiases[sumDims[dimsLength-1] + i] = expected[i] - result[i];
    }
    for(int layer = dimsLength-1; layer > 0; layer--) {
        // For every neuron in the layer
        for(int neuron = 0; neuron < dims[layer]; neuron++) {
            // For every neuron in the layer before
            for(int subNeuron = 0; subNeuron < dims[layer-1]; subNeuron++) {
                deltaWeights[sumWeights[layer-1] + dims[layer] * (subNeuron) + neuron] = values[sumDims[layer-1] + subNeuron] * deltaBiases[sumDims[layer] + neuron];
                deltaBiases[sumDims[layer-1] + subNeuron] += weights[sumWeights[layer-1] + dims[layer] * (subNeuron) + neuron] * deltaBiases[sumDims[layer] + neuron];
            }
        }
    }
}

void Network::backpropogateCPU(float expected[], float * result) {
    getBackPropDeltasCPU(expected, result);
    for(int neuron = 0; neuron < totalNNSize; neuron++) {
        biases[neuron] += deltaBiases[neuron] * learningRate;
    }
    for(int weight = 0; weight < totalWeights; weight++) {
        weights[weight] += deltaWeights[weight] * learningRate;
    }
}

void Network::unloadNetworkFromGPU() {
    hipFree(deviceValues);
    hipFree(deviceBiases);
    hipFree(deviceWeights);
    hipFree(deviceDims);
    hipFree(deviceSumDims);
    hipFree(deviceSumWeights);
    loaded = false;
};

Network::~Network() {
    free(biases);
    free(weights);
    free(dims);
    free(weightDims);
    free(sumDims);
    free(sumWeights);
    if(!loaded) {
        unloadNetworkFromGPU();
    }
};

void Network::print() {
    for(int layer = 0; layer < dimsLength; layer++) {
        std::cout << "Layer " << layer+1 << ":" << std::endl;
        for(int neuron = 0; neuron < dims[layer]; neuron++) {
            std::cout << "\tNeuron " << neuron+1 << ":" << std::endl;
            std::cout << "\t\tBias: " << biases[sumDims[layer] + neuron] << ":" << std::endl;
            std::cout << "\t\tDelta Bias: " << deltaBiases[sumDims[layer] + neuron] << ":" << std::endl;
            if(layer < (dimsLength-1)) {
                std::cout << "\t\tWeights: " << std::endl;
                for(int weight = 0; weight < dims[layer+1]; weight++) {
                    std::cout << "\t\t\tWeight " << weight+1 << ": " << weights[sumWeights[layer] + dims[layer+1]*neuron + weight] << ":" << std::endl;
                    std::cout << "\t\t\tDelta Weight " << weight+1 << ": " << deltaWeights[sumWeights[layer] + dims[layer+1]*neuron + weight] << ":" << std::endl;
                }
            }
        }
    }
};

void Network::save() {
    FILE * file = fopen("dimsLength", "wb");
    fwrite(&dimsLength, sizeof(int), 1, file);
    fclose(file);

    file = fopen("dims", "wb");
    fwrite(dims, sizeof(int), dimsLength, file);
    fclose(file);

    file = fopen("biases", "wb");
    fwrite(biases, sizeof(float), totalNNSize, file);
    fclose(file);

    file = fopen("weights", "wb");
    fwrite(weights, sizeof(float), totalWeights, file);
    fclose(file);
};

void Network::read() {
    //fread(data[i], sizeof(data[i][0]), ny, file);
    FILE * file = fopen("dimsLength", "rb");
    fread(&dimsLength, sizeof(int), 1, file);
    fclose(file);

    dims = (int *) malloc(dimsLength * sizeof(int));
    file = fopen("dims", "rb");
    fread(dims, sizeof(int), dimsLength, file);
    fclose(file);

    // This initializes the rest of the Network class so I can use totalNNSize and totalWeights in the next lines.
    initAll();
    
    file = fopen("biases", "rb");
    fread(biases, sizeof(float), totalNNSize, file);
    fclose(file);

    file = fopen("weights", "rb");
    fread(weights, sizeof(float), totalWeights, file);
    fclose(file);
};

int main() {
    int dims[] = {2, 3, 1};
    float * biases = (float *) calloc(6, sizeof(float));
    float * weights = (float *) calloc(9, sizeof(float));
    weights[0] = 0.25;
    weights[1] = 0.5;
    weights[2] = 0.75;
    weights[3] = 0.33;
    weights[4] = 0.66; 
    weights[5] = 1;
    weights[6] = 0.5;
    weights[7] = 0.5; 
    weights[8] = 0.5;
    Network test = Network(3, dims, 1);
    test.biases = biases;
    test.weights = weights;

    // Network test = Network(); // If no arguments are passed, it tries to read from a file.
    float passIn[] = {0.5, 1};
    test.loadNetworkToGPU();
    //test.print();
    //test.randomize(); // This does nothing to the result since the weights and biases were already loaded to the GPU.
    float * result = test.feedForwardCPU(passIn);
    float * result2 = test.feedForwardGPU(passIn);

    //test.unloadNetworkFromGPU(); // (Delete) should call unloadNetworkFromGPU, but I call it anyways.
    float expected[] = {1.5};
    test.getBackPropDeltasCPU(expected, result);
    test.backPropGPU(expected, result2);
    test.print();

    float * deltaWeights = (float *) malloc(test.totalWeights * sizeof(float));
    float * deltaBiases = (float *) malloc(test.totalNNSize * sizeof(float));
    hipMemcpy(deltaWeights, test.deviceDeltaWeights, test.totalWeights * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(deltaBiases, test.deviceDeltaBiases, test.totalNNSize * sizeof(float), hipMemcpyDeviceToHost);
    for(int i = 0; i < test.totalWeights; i++) {
        std::cout << "Weights: " << test.deltaWeights[i] << " " << deltaWeights[i] << " " <<  std::endl;
    }
    for(int i = 0; i < test.totalNNSize; i++) {
        std::cout << "Biases: " << test.deltaBiases[i] << " " << deltaBiases[i] << std::endl;
    }

    free(result);
    //test.save();
    std::cout << "Results: " << result[0] << std::endl;
    //std::cout << "Results: " << result2[0] << std::endl;
    // Expected outputs: 0.455, 0.91, 1.375, 1.37
}

/*
float abValue(float num) {
    return num > 0 ? num : -num;
}

int main() {
    int dims[] = {1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024};
    const int TIMESTOTEST = 100;
    const int INPUTLAYERAMOUNT = 1024;
    int dimsLength = sizeof(dims)/sizeof(dims[0]);
    float CPUSolutions[TIMESTOTEST];
    float GPUSolutions[TIMESTOTEST];
    Network test = Network(dimsLength, dims);
    test.randomize();
    test.loadNetworkToGPU();
    float input[TIMESTOTEST][INPUTLAYERAMOUNT];

    srand(23452345);

    for(int i = 0; i < TIMESTOTEST; i++) {
        for(int j = 0; j < INPUTLAYERAMOUNT; j++) {
            input[i][j] = 2*((float) rand() / (float) RAND_MAX) - 1;
        }
    }

    std::cout << "Starting CPU now." << std::endl;
    auto startCPU = std::chrono::high_resolution_clock::now();
    for(int i = 0; i < TIMESTOTEST; i++) {
        float * result = test.feedForwardCPU(input[i]);
        CPUSolutions[i] = result[0];
        free(result);
    }
    auto stopCPU = std::chrono::high_resolution_clock::now();

    std::cout << "Starting GPU now." << std::endl;
    auto startGPU = std::chrono::high_resolution_clock::now();
    for(int i = 0; i < TIMESTOTEST; i++) {
        float * result = test.feedForwardGPU(input[i]);
        GPUSolutions[i] = result[0];
        free(result);
    }
    auto stopGPU = std::chrono::high_resolution_clock::now();

    auto durationCPU = std::chrono::duration_cast<std::chrono::microseconds>(stopCPU - startCPU);
    auto durationGPU = std::chrono::duration_cast<std::chrono::microseconds>(stopGPU - startGPU);

    std::cout << "The CPU took " << durationCPU.count() << " microseconds." << std::endl;
    std::cout << "The GPU took " << durationGPU.count() << " microseconds." << std::endl;
    int correct = 0;
    for(int i = 0; i < TIMESTOTEST; i++) {
        correct += (abValue(CPUSolutions[i]/GPUSolutions[i]) > 0.95 && abValue(CPUSolutions[i]/GPUSolutions[i]) < 1.05);
        std::cout << GPUSolutions[i] << " " << CPUSolutions[i] << " " << abValue(CPUSolutions[i]/GPUSolutions[i]) << std::endl;
    }
    
    std::cout << "The GPU got " << correct << " correct answer(s)." << std::endl;
};*/
