#include "hip/hip_runtime.h"
#include<iostream>
#include<cstdlib>

__global__ void feedForwardLayer(float * values, float * biases, float * weights, int * dims, int * sumDims, int * sumWeights, int currentLayer) {
    /* This gets the 'index' of the thread. The thread id is the id of the thread in the thread block
       The blockDim is the amount of threads per block and the block id is the id of the block this thread is in
    */ This basically gives the index of the thread so you can use it to access memory.
    const int index = blockDim.x * blockIdx.x + threadIdx.x;

    // If the thread index is less than the size of the array.
    if(index < dims[currentLayer]) {
        float temp = 0;
        for(int subNeuron = sumDims[currentLayer-1]; subNeuron < dims[currentLayer]; subNeuron++) {
            temp += values[sumDims[currentLayer-1] + subNeuron] * weights[sumWeights[currentLayer-1] + dims[currentLayer]*subNeuron + index];
        }
        temp += biases[sumDims[currentLayer] + index];
        values[sumDims[currentLayer] + index] = temp/10; // This division by 10 is arbitrary, I'm using it to keep the numbers small.
    }
}


class Network {
    public:
        int THREADSPERBLOCK = 1024;
        bool loaded = false;
        float * biases;
        float * weights;
        int * dims;
        int * weightDims;
        int * sumDims;
        int * sumWeights;
        int dimsLength;
        int totalNNSize = 0;
        int totalWeights = 0;

        float * deviceBiases;
        float * deviceWeights;
        int * deviceDims;
        int * deviceSumDims;
        int * deviceSumWeights;        

        // Even though this could probably be made to run faster, this is branchless and clear.
        Network(int dimsLen, int * neuronCountList) {
            // Sets dims and dimsLength
            dimsLength = dimsLen;
            dims = (int *) malloc(dimsLength * sizeof(int));
            for(int layer = 0; layer < dimsLength; layer++) {
                dims[layer] = neuronCountList[layer];
            }

            // Gets the total size of the network.
            for(int layer = 0; layer < dimsLength; layer++) {
                totalNNSize += dims[layer];
            }

            // Gets the total amount of weights, and the amount of weights per layer
            weightDims = (int *) malloc((dimsLength) * sizeof(int));
            weightDims[dimsLength-1] = 0;
            for(int layer = 0; layer < dimsLength-1; layer++) {
                weightDims[layer] = dims[layer] * dims[layer+1];
                totalWeights += weightDims[layer];
            }

            // Gets the amount of weights from the 0th weight for each layer
            sumWeights = (int *) malloc((dimsLength) * sizeof(int));
            sumWeights[0] = 0;
            sumWeights[1] = weightDims[0];
            for(int layer = 1; layer < dimsLength-1; layer++) {
                sumWeights[layer+1] = weightDims[layer] + weightDims[layer+1];
            }

            // Gets the amount of neuron from the 0th neuron for each layer
            sumDims = (int *) malloc((dimsLength+1) * sizeof(int));
            sumDims[0] = 0;
            sumDims[1] = dims[0];
            for(int layer = 0; layer < dimsLength; layer++) {
                sumDims[layer+2] = sumDims[layer] + dims[layer+1];
            }

            // Initializes the weights and biases with a value of 0.
            biases = (float *) calloc(totalNNSize, sizeof(float));
            weights = (float *) calloc(totalWeights, sizeof(float));
        };
        void loadNetworkToGPU() {
            hipMalloc(&deviceBiases, totalNNSize * sizeof(float));
            hipMalloc(&deviceWeights, totalWeights * sizeof(float));
            hipMalloc(&deviceDims, dimsLength * sizeof(int));
            hipMalloc(&deviceSumDims, (dimsLength+1) * sizeof(int));
            hipMalloc(&deviceSumWeights, dimsLength * sizeof(int));

            hipMemcpy(deviceBiases, biases, totalNNSize * sizeof(float), hipMemcpyHostToDevice);
            hipMemcpy(deviceWeights, weights, totalWeights * sizeof(float), hipMemcpyHostToDevice);
            hipMemcpy(deviceDims, dims, dimsLength * sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(deviceSumDims, sumDims, (dimsLength+1) * sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(deviceSumWeights, sumWeights, dimsLength * sizeof(int), hipMemcpyHostToDevice);
            loaded = true;
        }
        float * feedForward(float input[]) {
            // Creates a new, empty value array.
            float * values = (float *) calloc(totalNNSize, sizeof(float));

            // Puts the inputs into the values to be fed forward.
            for(int neuron = 0; neuron < dims[0]; neuron++) {
                values[neuron] = input[neuron];
            }
            
            // Allocates device memory for the values
            float * deviceValues;
            hipMalloc(&deviceValues, totalNNSize * sizeof(float));
            hipMemcpy(deviceValues, values, totalNNSize * sizeof(float), hipMemcpyHostToDevice);

            // For each layer, feed forward using a GPU thread instead of a for loop.
            for(int layer = 1; layer < dimsLength; layer++) {
                /* Serial Method to feedforward
                for(int neuron = 0; neuron < dims[layer]; neuron++) {
                    for(int subNeuron = sumDims[layer-1]; subNeuron < sumDims[layer]; subNeuron++) {
                        values[sumDims[layer] + neuron] += values[sumDims[layer-1] + subNeuron] * weights[sumWeights[layer-1] + dims[layer]*subNeuron + neuron];
                    }
                }
                */
                int dimBlock = THREADSPERBLOCK;
                int dimGrid = (dims[layer] + THREADSPERBLOCK - 1)/THREADSPERBLOCK;
                feedForwardLayer <<< dimGrid, dimBlock >>> (deviceValues, deviceBiases, deviceWeights, deviceDims, deviceSumDims, deviceSumWeights, layer);
            }

            hipMemcpy(values, deviceValues, totalNNSize * sizeof(float), hipMemcpyDeviceToHost);
            hipFree(deviceValues);
            return values;
        }
        void unloadNetworkFromGPU() {
            hipFree(deviceBiases);
            hipFree(deviceWeights);
            hipFree(deviceDims);
            hipFree(deviceSumDims);
            hipFree(deviceSumWeights);
            loaded = false;
        }
        ~Network() {
            free(biases);
            free(weights);
            free(dims);
            free(weightDims);
            free(sumDims);
            free(sumWeights);
            if(!loaded) {
                unloadNetworkFromGPU();
            }
        };
        void print() {
            for(int layer = 0; layer < dimsLength; layer++) {
                std::cout << "Layer " << layer+1 << ":" << std::endl;
                for(int neuron = 0; neuron < dims[layer]; neuron++) {
                    std::cout << "\tNeuron " << neuron+1 << ":" << std::endl;
                    std::cout << "\t\tBias: " << biases[sumDims[layer] + neuron] << ":" << std::endl;
                    if(layer < (dimsLength-1)) {
                        std::cout << "\t\tWeights: " << std::endl;
                        for(int weight = 0; weight < dims[layer+1]; weight++) {
                            std::cout << "\t\t\tWeight " << weight+1 << ": " << weights[sumWeights[layer] + dims[layer+1]*neuron + weight] << ":" << std::endl;
                        }
                    }
                }
            }
        }
};

/* Some sample code:

int main() {
    int dims[] = {2, 3};
    float * biases = (float *) calloc(5, sizeof(float));
    float * weights = (float *) calloc(6, sizeof(float));
    weights[0] = 0.25;
    weights[1] = 0.5;
    weights[2] = 0.75;
    weights[3] = 0.33;
    weights[4] = 0.66; 
    weights[5] = 1;
    Network * test = new Network(2, dims);
    test.biases = biases;
    test.weights = weights;
    float passIn[] = {0.5, 1};
    test.loadNetworkToGPU();
    float * result = test.feedForward(passIn);
    test.unloadNetworkFromGPU();
    delete test; // Delete should call unloadNetworkFromGPU, but I call it anyways.
    test.print();
    std::cout << "Results: " << result[2] << " " << result[3] << " " << result[4] << std::endl;
    std::cout << result[2] << std::endl;
    std::cout << result[3] << std::endl;
    std::cout << result[4] << std::endl;
    // Expected outputs: 0.455, 0.91, 1.375
}
*/
